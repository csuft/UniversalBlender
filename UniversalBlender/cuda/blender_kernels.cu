#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_texture_types.h>
#include <>
#include <algorithm>
#include <cmath> 
#include <vector>
#include <chrono>
#include <memory>
#include <fstream> 
#include <time.h>  
#include "../utils/log.h"

hipError_t checkError(hipError_t ret) {
	if (ret != hipSuccess) {
		LOGERR("cuda err:%s ,file:%s,line:%d ...", hipGetErrorString(ret), __FILE__, __LINE__);
		return ret;
	}
}

texture<uchar4, hipTextureType2D, hipReadModeNormalizedFloat> tex;

//kernel function
__global__ void mapFinishToBlender(int blend_width, int image_width, float *left_map, float *right_map, float *alpha_table, unsigned char *out_img)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = y * image_width + x;
	int left_start = image_width >> 2;
	float *location = nullptr;

	if (x < left_start - blend_width)
	{
		location = left_map + index * 2;
		goto MAP;
	}
	else if (x >= left_start - blend_width && x < left_start + blend_width)
	{
		goto BLEND_LEFT;
	}
	else if (x >= left_start + blend_width && x < left_start * 3 - blend_width)
	{
		location = right_map + index * 2;
		goto MAP;
	}
	else if (x >= left_start * 3 - blend_width && x < left_start * 3 + blend_width)
	{
		goto BLEND_RIGHT;
	}
	else if (x >= left_start * 3 + blend_width)
	{
		location = left_map + index * 2;
		goto MAP;
	}

MAP:
	{
		float4 val = tex2D(tex, location[0], location[1]);
		out_img[4 * index + 0] = val.x * 255;
		out_img[4 * index + 1] = val.y * 255;
		out_img[4 * index + 2] = val.z * 255;
		out_img[4 * index + 3] = 255;
		return;
	}

BLEND_LEFT:
	{
		float *location1 = nullptr;
		int plane_left = left_start - blend_width;
		float alpha = *(alpha_table + 2 * blend_width - (x - plane_left) - 1);

		location = left_map + index * 2;
		location1 = right_map + index * 2;
		float4 val0 = tex2D(tex, location[0], location[1]);
		float4 val1 = tex2D(tex, location1[0], location1[1]);

		out_img[4 * index + 0] = (val0.x*alpha + val1.x*(1 - alpha)) * 255;
		out_img[4 * index + 1] = (val0.y*alpha + val1.y*(1 - alpha)) * 255;
		out_img[4 * index + 2] = (val0.z*alpha + val1.z*(1 - alpha)) * 255;
		out_img[4 * index + 3] = 255;
		return;
	}

BLEND_RIGHT:
	{
		float *location1 = nullptr;
		int plane_left = left_start * 3 - blend_width;
		float alpha = *(alpha_table + (x - plane_left));

		location = left_map + index * 2;
		location1 = right_map + index * 2;
		float4 val0 = tex2D(tex, location[0], location[1]);
		float4 val1 = tex2D(tex, location1[0], location1[1]);

		out_img[4 * index + 0] = (val0.x*alpha + val1.x*(1 - alpha)) * 255;
		out_img[4 * index + 1] = (val0.y*alpha + val1.y*(1 - alpha)) * 255;
		out_img[4 * index + 2] = (val0.z*alpha + val1.z*(1 - alpha)) * 255;
		out_img[4 * index + 3] = 255;
		return;
	}


}
//cuda  blender

extern "C" hipError_t cuFinishToBlender(hipArray *inputBuffer, float *left_map, float*right_map, float* alpha_table, int image_width, int image_height, int bd_width, dim3 thread, dim3 numBlock, unsigned char *uOutBuffer)
{
	hipError_t ret = hipSuccess;
	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.normalized = false;
	tex.filterMode = hipFilterModeLinear;

	hipChannelFormatDesc channelDesc; 
	checkError(hipBindTextureToArray(tex, inputBuffer));
	mapFinishToBlender <<<numBlock, thread >>>(bd_width, image_width, left_map, right_map, alpha_table, uOutBuffer);

	return ret;

}

// Convert RGB(BGR) to RGBA(BGRA)
__global__ void add_alpha_channel(unsigned char* input, unsigned char* output)
{

}

// Convert RGBA(BGRA) to RGB(BGR)
__global__ void remove_alpha_channel(unsigned char* input, unsigned char* output)
{

}
